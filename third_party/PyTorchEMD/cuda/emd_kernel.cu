#include "hip/hip_runtime.h"
/**********************************
 * Original Author: Haoqiang Fan
 * Modified by: Kaichun Mo
 *********************************/

#ifndef _EMD_KERNEL
#define _EMD_KERNEL

#include <cmath>
#include <vector>

#include <ATen/ATen.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>  // at::cuda::getApplyGrid

#define CHECK_INPUT(x) 


/********************************
* Forward kernel for approxmatch
*********************************/

template<typename scalar_t>
__global__ void approxmatch(int b,int n,int m,const scalar_t * __restrict__ xyz1,const scalar_t * __restrict__ xyz2,scalar_t * __restrict__ match,scalar_t * temp){
	scalar_t * remainL=temp+blockIdx.x*(n+m)*2, * remainR=temp+blockIdx.x*(n+m)*2+n,*ratioL=temp+blockIdx.x*(n+m)*2+n+m,*ratioR=temp+blockIdx.x*(n+m)*2+n+m+n;
	scalar_t multiL,multiR;
	if (n>=m){
		multiL=1;
		multiR=n/m;
	}else{
		multiL=m/n;
		multiR=1;
	}
	const int Block=1024;
	__shared__ scalar_t buf[Block*4];
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		for (int j=threadIdx.x;j<n*m;j+=blockDim.x)
			match[i*n*m+j]=0;
		for (int j=threadIdx.x;j<n;j+=blockDim.x)
			remainL[j]=multiL;
		for (int j=threadIdx.x;j<m;j+=blockDim.x)
			remainR[j]=multiR;
		__syncthreads();
		for (int j=7;j>=-2;j--){
			scalar_t level=-powf(4.0f,j);
			if (j==-2){
				level=0;
			}
			for (int k0=0;k0<n;k0+=blockDim.x){
				int k=k0+threadIdx.x;
				scalar_t x1=0,y1=0,z1=0;
				if (k<n){
					x1=xyz1[i*n*3+k*3+0];
					y1=xyz1[i*n*3+k*3+1];
					z1=xyz1[i*n*3+k*3+2];
				}
				scalar_t suml=1e-9f;
				for (int l0=0;l0<m;l0+=Block){
					int lend=min(m,l0+Block)-l0;
					for (int l=threadIdx.x;l<lend;l+=blockDim.x){
						scalar_t x2=xyz2[i*m*3+l0*3+l*3+0];
						scalar_t y2=xyz2[i*m*3+l0*3+l*3+1];
						scalar_t z2=xyz2[i*m*3+l0*3+l*3+2];
						buf[l*4+0]=x2;
						buf[l*4+1]=y2;
						buf[l*4+2]=z2;
						buf[l*4+3]=remainR[l0+l];
					}
					__syncthreads();
					for (int l=0;l<lend;l++){
						scalar_t x2=buf[l*4+0];
						scalar_t y2=buf[l*4+1];
						scalar_t z2=buf[l*4+2];
						scalar_t d=level*((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1));
						scalar_t w=__expf(d)*buf[l*4+3];
						suml+=w;
					}
					__syncthreads();
				}
				if (k<n)
					ratioL[k]=remainL[k]/suml;
			}
			__syncthreads();
			for (int l0=0;l0<m;l0+=blockDim.x){
				int l=l0+threadIdx.x;
				scalar_t x2=0,y2=0,z2=0;
				if (l<m){
					x2=xyz2[i*m*3+l*3+0];
					y2=xyz2[i*m*3+l*3+1];
					z2=xyz2[i*m*3+l*3+2];
				}
				scalar_t sumr=0;
				for (int k0=0;k0<n;k0+=Block){
					int kend=min(n,k0+Block)-k0;
					for (int k=threadIdx.x;k<kend;k+=blockDim.x){
						buf[k*4+0]=xyz1[i*n*3+k0*3+k*3+0];
						buf[k*4+1]=xyz1[i*n*3+k0*3+k*3+1];
						buf[k*4+2]=xyz1[i*n*3+k0*3+k*3+2];
						buf[k*4+3]=ratioL[k0+k];
					}
					__syncthreads();
					for (int k=0;k<kend;k++){
						scalar_t x1=buf[k*4+0];
						scalar_t y1=buf[k*4+1];
						scalar_t z1=buf[k*4+2];
						scalar_t w=__expf(level*((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1)))*buf[k*4+3];
						sumr+=w;
					}
					__syncthreads();
				}
				if (l<m){
					sumr*=remainR[l];
					scalar_t consumption=fminf(remainR[l]/(sumr+1e-9f),1.0f);
					ratioR[l]=consumption*remainR[l];
					remainR[l]=fmaxf(0.0f,remainR[l]-sumr);
				}
			}
			__syncthreads();
			for (int k0=0;k0<n;k0+=blockDim.x){
				int k=k0+threadIdx.x;
				scalar_t x1=0,y1=0,z1=0;
				if (k<n){
					x1=xyz1[i*n*3+k*3+0];
					y1=xyz1[i*n*3+k*3+1];
					z1=xyz1[i*n*3+k*3+2];
				}
				scalar_t suml=0;
				for (int l0=0;l0<m;l0+=Block){
					int lend=min(m,l0+Block)-l0;
					for (int l=threadIdx.x;l<lend;l+=blockDim.x){
						buf[l*4+0]=xyz2[i*m*3+l0*3+l*3+0];
						buf[l*4+1]=xyz2[i*m*3+l0*3+l*3+1];
						buf[l*4+2]=xyz2[i*m*3+l0*3+l*3+2];
						buf[l*4+3]=ratioR[l0+l];
					}
					__syncthreads();
					scalar_t rl=ratioL[k];
					if (k<n){
						for (int l=0;l<lend;l++){
							scalar_t x2=buf[l*4+0];
							scalar_t y2=buf[l*4+1];
							scalar_t z2=buf[l*4+2];
							scalar_t w=__expf(level*((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1)))*rl*buf[l*4+3];
							match[i*n*m+(l0+l)*n+k]+=w;
							suml+=w;
						}
					}
					__syncthreads();
				}
				if (k<n)
					remainL[k]=fmaxf(0.0f,remainL[k]-suml);
			}
			__syncthreads();
		}
	}
}

//void approxmatchLauncher(int b,int n,int m,const scalar_t * xyz1,const scalar_t * xyz2,scalar_t * match,scalar_t * temp){
//	approxmatch<<<32,512>>>(b,n,m,xyz1,xyz2,match,temp);
//}

/* ApproxMatch forward interface
Input:
  xyz1: (B, N1, 3)  # dataset_points
  xyz2: (B, N2, 3)  # query_points
Output:
  match: (B, N2, N1)
*/
at::Tensor ApproxMatchForward(
    const at::Tensor xyz1,
    const at::Tensor xyz2){
  const auto b = xyz1.size(0);
  const auto n = xyz1.size(1);
  const auto m = xyz2.size(1);

  TORCH_CHECK_EQ(xyz2.size(0), b);
  TORCH_CHECK_EQ(xyz1.size(2), 3);
  TORCH_CHECK_EQ(xyz2.size(2), 3);
  CHECK_INPUT(xyz1);
  CHECK_INPUT(xyz2);

  auto match = at::zeros({b, m, n}, xyz1.type());
  auto temp = at::zeros({b, (n+m)*2}, xyz1.type());

  AT_DISPATCH_FLOATING_TYPES(xyz1.scalar_type(), "ApproxMatchForward", ([&] {
        approxmatch<scalar_t><<<32,512>>>(b, n, m, xyz1.data<scalar_t>(), xyz2.data<scalar_t>(), match.data<scalar_t>(), temp.data<scalar_t>());
  }));
  AT_CUDA_CHECK(hipGetLastError());

  return match;
}


/********************************
* Forward kernel for matchcost
*********************************/

template<typename scalar_t>
__global__ void matchcost(int b,int n,int m,const scalar_t * __restrict__ xyz1,const scalar_t * __restrict__ xyz2,const scalar_t * __restrict__ match,scalar_t * __restrict__ out){
	__shared__ scalar_t allsum[512];
	const int Block=1024;
	__shared__ scalar_t buf[Block*3];
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		scalar_t subsum=0;
		for (int k0=0;k0<n;k0+=blockDim.x){
			int k=k0+threadIdx.x;
			scalar_t x1=0,y1=0,z1=0;
			if (k<n){
				x1=xyz1[i*n*3+k*3+0];
				y1=xyz1[i*n*3+k*3+1];
				z1=xyz1[i*n*3+k*3+2];
			}
			for (int l0=0;l0<m;l0+=Block){
				int lend=min(m,l0+Block)-l0;
				for (int l=threadIdx.x;l<lend*3;l+=blockDim.x)
					buf[l]=xyz2[i*m*3+l0*3+l];
				__syncthreads();
				if (k<n){
					for (int l=0;l<lend;l++){
						scalar_t x2=buf[l*3+0];
						scalar_t y2=buf[l*3+1];
						scalar_t z2=buf[l*3+2];
						scalar_t d=(x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1);
						subsum+=d*match[i*n*m+(l0+l)*n+k];
					}
				}
				__syncthreads();
			}
		}
		allsum[threadIdx.x]=subsum;
		for (int j=1;j<blockDim.x;j<<=1){
			__syncthreads();
			if ((threadIdx.x&j)==0 && threadIdx.x+j<blockDim.x){
				allsum[threadIdx.x]+=allsum[threadIdx.x+j];
			}
		}
		if (threadIdx.x==0)
			out[i]=allsum[0];
		__syncthreads();
	}
}

//void matchcostLauncher(int b,int n,int m,const scalar_t * xyz1,const scalar_t * xyz2,const scalar_t * match,scalar_t * out){
//	matchcost<<<32,512>>>(b,n,m,xyz1,xyz2,match,out);
//}

/* MatchCost forward interface
Input:
  xyz1: (B, N1, 3)  # dataset_points
  xyz2: (B, N2, 3)  # query_points
  match: (B, N2, N1)
Output:
  cost: (B)
*/
at::Tensor MatchCostForward(
    const at::Tensor xyz1,
    const at::Tensor xyz2,
    const at::Tensor match){
  const auto b = xyz1.size(0);
  const auto n = xyz1.size(1);
  const auto m = xyz2.size(1);

  TORCH_CHECK_EQ(xyz2.size(0), b);
  TORCH_CHECK_EQ(xyz1.size(2), 3);
  TORCH_CHECK_EQ(xyz2.size(2), 3);
  CHECK_INPUT(xyz1);
  CHECK_INPUT(xyz2);

  auto cost = at::zeros({b}, xyz1.type());

  AT_DISPATCH_FLOATING_TYPES(xyz1.scalar_type(), "MatchCostForward", ([&] {
        matchcost<scalar_t><<<32,512>>>(b, n, m, xyz1.data<scalar_t>(), xyz2.data<scalar_t>(), match.data<scalar_t>(), cost.data<scalar_t>());
  }));
  AT_CUDA_CHECK(hipGetLastError());

  return cost;
}


/********************************
* matchcostgrad2 kernel
*********************************/

template<typename scalar_t>
__global__ void matchcostgrad2(int b,int n,int m,const scalar_t * __restrict__ grad_cost,const scalar_t * __restrict__ xyz1,const scalar_t * __restrict__ xyz2,const scalar_t * __restrict__ match,scalar_t * __restrict__ grad2){
	__shared__ scalar_t sum_grad[256*3];
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		int kbeg=m*blockIdx.y/gridDim.y;
		int kend=m*(blockIdx.y+1)/gridDim.y;
		for (int k=kbeg;k<kend;k++){
			scalar_t x2=xyz2[(i*m+k)*3+0];
			scalar_t y2=xyz2[(i*m+k)*3+1];
			scalar_t z2=xyz2[(i*m+k)*3+2];
			scalar_t subsumx=0,subsumy=0,subsumz=0;
			for (int j=threadIdx.x;j<n;j+=blockDim.x){
				scalar_t x1=x2-xyz1[(i*n+j)*3+0];
				scalar_t y1=y2-xyz1[(i*n+j)*3+1];
				scalar_t z1=z2-xyz1[(i*n+j)*3+2];
				scalar_t d=match[i*n*m+k*n+j]*2;
				subsumx+=x1*d;
				subsumy+=y1*d;
				subsumz+=z1*d;
			}
			sum_grad[threadIdx.x*3+0]=subsumx;
			sum_grad[threadIdx.x*3+1]=subsumy;
			sum_grad[threadIdx.x*3+2]=subsumz;
			for (int j=1;j<blockDim.x;j<<=1){
				__syncthreads();
				int j1=threadIdx.x;
				int j2=threadIdx.x+j;
				if ((j1&j)==0 && j2<blockDim.x){
					sum_grad[j1*3+0]+=sum_grad[j2*3+0];
					sum_grad[j1*3+1]+=sum_grad[j2*3+1];
					sum_grad[j1*3+2]+=sum_grad[j2*3+2];
				}
			}
			if (threadIdx.x==0){
				grad2[(i*m+k)*3+0]=sum_grad[0]*grad_cost[i];
				grad2[(i*m+k)*3+1]=sum_grad[1]*grad_cost[i];
				grad2[(i*m+k)*3+2]=sum_grad[2]*grad_cost[i];
			}
			__syncthreads();
		}
	}
}

/********************************
* matchcostgrad1 kernel
*********************************/

template<typename scalar_t>
__global__ void matchcostgrad1(int b,int n,int m,const scalar_t * __restrict__ grad_cost,const scalar_t * __restrict__ xyz1,const scalar_t * __restrict__ xyz2,const scalar_t * __restrict__ match,scalar_t * __restrict__ grad1){
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		for (int l=threadIdx.x;l<n;l+=blockDim.x){
			scalar_t x1=xyz1[i*n*3+l*3+0];
			scalar_t y1=xyz1[i*n*3+l*3+1];
			scalar_t z1=xyz1[i*n*3+l*3+2];
			scalar_t dx=0,dy=0,dz=0;
			for (int k=0;k<m;k++){
				scalar_t x2=xyz2[i*m*3+k*3+0];
				scalar_t y2=xyz2[i*m*3+k*3+1];
				scalar_t z2=xyz2[i*m*3+k*3+2];
				scalar_t d=match[i*n*m+k*n+l]*2;
				dx+=(x1-x2)*d;
				dy+=(y1-y2)*d;
				dz+=(z1-z2)*d;
			}
			grad1[i*n*3+l*3+0]=dx*grad_cost[i];
			grad1[i*n*3+l*3+1]=dy*grad_cost[i];
			grad1[i*n*3+l*3+2]=dz*grad_cost[i];
		}
	}
}

//void matchcostgradLauncher(int b,int n,int m,const scalar_t * xyz1,const scalar_t * xyz2,const scalar_t * match,scalar_t * grad1,scalar_t * grad2){
//	matchcostgrad1<<<32,512>>>(b,n,m,xyz1,xyz2,match,grad1);
//	matchcostgrad2<<<dim3(32,32),256>>>(b,n,m,xyz1,xyz2,match,grad2);
//}


/* MatchCost backward interface
Input:
  grad_cost: (B)    # gradients on cost
  xyz1: (B, N1, 3)  # dataset_points
  xyz2: (B, N2, 3)  # query_points
  match: (B, N2, N1)
Output:
  grad1: (B, N1, 3)
  grad2: (B, N2, 3)
*/
std::vector<at::Tensor> MatchCostBackward(
    const at::Tensor grad_cost,
    const at::Tensor xyz1,
    const at::Tensor xyz2,
    const at::Tensor match){
  const auto b = xyz1.size(0);
  const auto n = xyz1.size(1);
  const auto m = xyz2.size(1);

  TORCH_CHECK_EQ(xyz2.size(0), b);
  TORCH_CHECK_EQ(xyz1.size(2), 3);
  TORCH_CHECK_EQ(xyz2.size(2), 3);
  CHECK_INPUT(xyz1);
  CHECK_INPUT(xyz2);

  auto grad1 = at::zeros({b, n, 3}, xyz1.type());
  auto grad2 = at::zeros({b, m, 3}, xyz1.type());

  AT_DISPATCH_FLOATING_TYPES(xyz1.scalar_type(), "MatchCostBackward", ([&] {
        matchcostgrad1<scalar_t><<<32,512>>>(b, n, m, grad_cost.data<scalar_t>(), xyz1.data<scalar_t>(), xyz2.data<scalar_t>(), match.data<scalar_t>(), grad1.data<scalar_t>());
        matchcostgrad2<scalar_t><<<dim3(32,32),256>>>(b, n, m, grad_cost.data<scalar_t>(), xyz1.data<scalar_t>(), xyz2.data<scalar_t>(), match.data<scalar_t>(), grad2.data<scalar_t>());
  }));
  AT_CUDA_CHECK(hipGetLastError());

  return std::vector<at::Tensor>({grad1, grad2});
}

#endif
